#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_complex.h>
#include <omp.h>
#include <dirent.h>
#include <unistd.h>
#include <errno.h>
#include <fcntl.h>
#include <sys/wait.h>
#include "defaults.h"
#include <hiprand/hiprand_kernel.h>

int main(){
	int q1, q2, M = M0, MTH = MTH0, L = 2 * NN + 1, blocks = len_beta;
	double p8 = p80, Y = (2 * pi) / 3 * p8 * p8, t0, t = omp_get_wtime(), a = 2.0 * pi / L;
	printf("L=%d, N=%d, block size=%d, number of beta=%d, sigma=%.7lf\n", NN, L-1, block_size, len_beta, sigma0);
	printf("dc=%d, M=%d, MTH=%d, threads=%d\n", dc, M, MTH, omp_get_num_procs());
	setbuf(stdout, NULL);

	unsigned long long seed = time(NULL);

	double *device_beta;
	hipMalloc(&device_beta, L * L * len_beta * sizeof(double));
	hipDoubleComplex *host_h = (hipDoubleComplex *)malloc(L * L * dc * len_beta * sizeof(hipDoubleComplex));
	hipDoubleComplex *device_h;
	hipMalloc(&device_h, L * L * dc * len_beta * sizeof(hipDoubleComplex));

	hipDoubleComplex *device_S;
	hipMalloc(&device_S, L * L * len_beta * sizeof(hipDoubleComplex));
	
	hipDoubleComplex *dump_h = (hipDoubleComplex*)malloc(L * L * sizeof(hipDoubleComplex));
	double *dump_g = (double*)malloc(L * L * sizeof(double));
	int *dump_c = (int*)malloc(2 * L * L * sizeof(int));
	
	double *device_dS_re;
	hipMalloc(&device_dS_re, L * L * len_beta * sizeof(double));

	double *device_dS_im;
	hipMalloc(&device_dS_im, L * L * len_beta * sizeof(double));

	double *host_g = (double *)malloc(L * L * dc * len_beta * sizeof(double));
	double *device_g;
	hipMalloc(&device_g, L * L * dc * len_beta * sizeof(double));

	int *host_c = (int *)malloc(2 * L * L * len_beta * sizeof(int));
	int *device_c;
	hipMalloc(&device_c, 2 * L * L * len_beta * sizeof(int));

	double *host_sines = (double *)malloc(L * sizeof(double));
	double *device_sines;
	hipMalloc(&device_sines, L * sizeof(double));

	double *host_Q = (double *)malloc(L * L * sizeof(double));
	double *device_Q;
	hipMalloc(&device_Q, L * L * sizeof(double));

	double *device_F_all;
	hipMalloc(&device_F_all, L * L * len_beta * sizeof(double));

	double *device_av_h;
	hipMalloc(&device_av_h, L * L * dc * len_beta * sizeof(double));
	hipMemset(device_av_h, 0, L * L * dc * len_beta * sizeof(double));

	double *host_av_cor_cor = (double *)malloc(L * L * dc * len_beta * sizeof(double));
	double *device_av_cor_cor;
	hipMalloc(&device_av_cor_cor, L * L * dc * len_beta * sizeof(double));
	hipMemset(device_av_cor_cor, 0, L * L * dc * len_beta * sizeof(double));

	double *host_av_h_h = (double *)malloc(L * L * dc * len_beta * sizeof(double));
	double *device_av_h_h;
	hipMalloc(&device_av_h_h, L * L * dc * len_beta * sizeof(double));
	hipMemset(device_av_h_h, 0, L * L * dc * len_beta * sizeof(double));

	double *host_av_av_cor_cor = (double *)calloc(L * L * dc, sizeof(double));
	double *host_av_av_h_h = (double *)calloc(L * L * dc, sizeof(double));

	#pragma omp parallel for
	for (q1 = -NN; q1 < NN+1; q1++) {
		host_sines[(q1 + L) % L] = sin(a * q1);
	}
	#pragma omp parallel for collapse(2)
	for (q1 = -NN; q1 < NN + 1; q1++) {
		for (q2 = -NN; q2 < NN + 1; q2++) {
			host_Q[(((q1 + L) % L) * L)+((q2 + L) % L)] = 4 * (sin(a * q1 / 2) * sin(a * q1 / 2) + sin(a * q2 / 2) * sin(a * q2 / 2));
		}
	}
	hipMemcpy(device_sines, host_sines, L * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_Q, host_Q, L * L * sizeof(double), hipMemcpyHostToDevice);
	printf("initialization\n");
	
	printf("generating noise\n");
	hipDeviceSynchronize();
	generate_noise<<<blocks, block_size>>>(device_beta, seed);
	printf("noise generated\n");
	hipDeviceSynchronize();
	
	
	if (!init(host_h, host_g, host_c)) {
		printf("no previous data\n");
		#pragma omp parallel for collapse(2)
		for (int i = 0; i < dc; i++) {
			for (int j = 0; j < len_beta; j++) {
				host_h[j * L * L * dc + i * L * L] = make_hipDoubleComplex(1 / (a * a), 0.0);
			}
		}
		#pragma omp parallel for collapse(2)
		for (int q1 = 0; q1 < L; q1++) {
			for(int q2 = 0; q2 < L; q2++) {
				if (!q1 && !q2) continue;
				for (int j = 0; j < len_beta; j++) {
					host_c[2 * j * L * L + 0 * L * L + q1 * L + q2] = 0;
					host_c[2 * j * L * L + 1 * L * L + q1 * L + q2] = 1;
					for (int i = 0; i < dc; i++) {
						host_h[j * L * L * dc + i * L * L + q1 * L + q2] = make_hipDoubleComplex(1.0 / host_Q[q1 * L + q2], 0);
						host_g[j * L * L * dc + i * L * L + q1 * L + q2] = hipCreal(hipCmul(host_h[j * L * L * dc + i * L * L + q1 * L + q2], host_h[j * L * L * dc + i * L * L + q1 * L + q2]));
					}
				}
			}
		}
		hipMemcpy(device_h, host_h, L * L * dc * len_beta * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
		hipMemcpy(device_g, host_g, L * L * dc * len_beta * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(device_c, host_c, 2 * L * L * len_beta * sizeof(int), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		printf("initial S calculating\n");
		calcS<<<blocks, block_size>>>(device_h, device_S, device_sines, device_Q);
		hipDeviceSynchronize();

		t0 = omp_get_wtime();
		printf("thermalization stage\n");
		hipDeviceSynchronize();
		simulate<<<blocks, block_size>>>(MTH, device_h, device_S, device_beta, NULL, NULL, Y, device_dS_re, device_dS_im, device_sines, device_Q, seed, device_F_all, device_av_h, device_av_cor_cor, device_av_h_h);
		hipDeviceSynchronize();
		printf("thermalization time: \t%.2lf min\n", (omp_get_wtime()-t0)/60);
	}
	
	else {
		printf("previous avereged data is loaded\n");
		printf("initial S calculating\n");
		hipMemcpy(device_h, host_h, L * L * dc * len_beta * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
		hipMemcpy(device_g, host_g, L * L * dc * len_beta * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(device_c, host_c, 2 * L * L * len_beta * sizeof(int), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		calcS<<<blocks, block_size>>>(device_h, device_S, device_sines, device_Q);
		hipDeviceSynchronize();
	}

	t0 = omp_get_wtime();
	printf("simulation stage\n");
	hipDeviceSynchronize();
	simulate<<<blocks, block_size>>>(M, device_h, device_S, device_beta, device_g, device_c, Y, device_dS_re, device_dS_im, device_sines, device_Q, seed, device_F_all, device_av_h, device_av_cor_cor, device_av_h_h);
	hipDeviceSynchronize();

	hipFree(device_S);
	hipFree(device_dS_re);
	hipFree(device_dS_im);
	free(host_sines);
	hipFree(device_sines);
	free(host_Q);
	hipFree(device_Q);


	hipMemcpy(host_h, device_h, L * L * dc * len_beta * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	hipMemcpy(host_g, device_g, L * L * dc * len_beta * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(host_c, device_c, 2 * L * L * len_beta * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(host_av_cor_cor, device_av_cor_cor, L * L * dc * len_beta * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(host_av_h_h, device_av_h_h, L * L * dc * len_beta * sizeof(double), hipMemcpyDeviceToHost);

	printf("simulation time: \t%.2lf min\n", (omp_get_wtime()-t0)/60);

	printf("saving data\n");
	hipDeviceSynchronize();
	dump(host_g, host_c, host_av_av_cor_cor, host_av_av_h_h, host_av_cor_cor, host_av_h_h, host_h, dump_h, dump_g, dump_c);
	hipFree(host_g);
	hipFree(device_g);
	hipFree(host_c);
	hipFree(device_c);
	free(host_h);
	hipFree(device_h);
	hipDeviceReset();
	printf("total elapsed time: \t%.2lf min\n", (omp_get_wtime()-t)/60);
}
